#include <iostream>
#include "hip/hip_runtime.h"

void stupidfunction() {
	float* a = (float*)malloc(100000 * sizeof(float));
}
int main(int argc, char* argv[]){

	hipError_t error;
	hipDeviceProp_t prop;
	int count;																//stores the number of CUDA compatible devices

	error = hipGetDeviceCount(&count);										//get the number of devices with compute capability < 1.0

	if(error != hipSuccess){												//if there is an error getting the device count
		std::cout<<"ERROR calling hipGetDeviceCount()"<<std::endl;			//display an error message
		return error;														//return the error
	}

	std::cout<<"Number of CUDA devices: "<<count<<std::endl;
	std::cout<<"Device 0 Properties-------------------------"<<std::endl;
	
	error = hipGetDeviceProperties(&prop, 0);								//get the properties for the first CUDA device

	if(error != hipSuccess){												//if there is an error getting the device properties
		std::cout<<"ERROR calling hipGetDeviceProperties()"<<std::endl;	//display an error message
		return error;														//return the error
	}

	stupidfunction();

	std::cout<<"Name:                  "<<prop.name<<std::endl
			 <<"Global Memory:         "<<(double)prop.totalGlobalMem/1024/1000000<<" Gb"<<std::endl
			 <<"Shared Memory/block:   "<<(double)prop.sharedMemPerBlock/1024<<" Kb"<<std::endl
			 <<"Registers/block:       "<<prop.regsPerBlock<<std::endl
			 <<"Warp Size:             "<<prop.warpSize<<std::endl
			 <<"Max Threads/block:     "<<prop.maxThreadsPerBlock<<std::endl
			 <<"Max Block Dimensions:  ["
			 						  <<prop.maxThreadsDim[0]<<" x "
			 						  <<prop.maxThreadsDim[1]<<" x "
			 						  <<prop.maxThreadsDim[2]<<"]"<<std::endl
			 <<"Max Grid Dimensions:   ["
			 						  <<prop.maxGridSize[0]<<" x "
			 						  <<prop.maxGridSize[1]<<" x "
			 						  <<prop.maxGridSize[2]<<"]"<<std::endl
			 <<"Constant Memory:       "<<(double)prop.totalConstMem/1024<<" Kb"<<std::endl
			 <<"Compute Capability:    "<<prop.major<<"."<<prop.minor<<std::endl
			 <<"Clock Rate:            "<<(double)prop.clockRate/1000000<<" GHz"<<std::endl;
}